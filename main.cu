#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <memory>
#include <string> 

using namespace cv;
using namespace std;

#define HANDLE_ERROR(ans) { handleCudaError((ans), __FILE__, __LINE__); }
inline void handleCudaError(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"[ERROR]: %s in %s:%d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


struct Coords
{
    int width;
    int heigth;
};

class ImageManager
{
public:
    ImageManager() = default;
    ~ImageManager() = default;

    Coords readImage(string imagePath);  
    void saveImage(string outputPath);
    void setBlockSize(int blockSize);
    void setGridSize(int gridSize);
    int getBlockSize();
    int getGridSize();
    void setStartPixel(Coords startCoords);
    void setTreshold(int treshold);
    void clearRunningTime();
    double getRunningTime();
    void performWatershade();
private:
    void prepareOutputImage(Mat& inputImage);

    Mat inputImage;
    Mat outputImage;
    int blockSize;
    int gridSize;
    Coords startPixel;
    int treshold;
    float runningTime;
};

struct RGB
{
    int red;
    int green;
    int blue;
};

__global__ void makeGrayscaleImage(unsigned char* imageIn, unsigned char* imageOut, long xSize, long ySize)
{
    long xCoord = blockIdx.x * blockDim.x + threadIdx.x;
    long yCoord = blockIdx.y * blockDim.y + threadIdx.y;
    
    double redWeigth = 0.33;
    double greenWeigth = 0.34;
    double blueWeigth = 0.33;

    double currentColor;
    int inputIndex;
    for(long x=xCoord; x < xSize; x+=(blockDim.x * gridDim.x))
    {
    	for(long y=yCoord; y < ySize; y+=(blockDim.y * gridDim.y))
        {
            currentColor = 0.0;
			inputIndex = xSize * y * 3 + x * 3;

			currentColor += imageIn[inputIndex + 2] * redWeigth;
			currentColor += imageIn[inputIndex + 1] * greenWeigth;
			currentColor += imageIn[inputIndex] * blueWeigth;
            imageOut[xSize * y + x] = (unsigned char)currentColor;
	    }
    }
}

__global__ void makeDifferenceImage(unsigned char* grayscaleImage, unsigned char* differenceImage, long xSize, long ySize)
{
    long xCoord = blockIdx.x * blockDim.x + threadIdx.x;
    long yCoord = blockIdx.y * blockDim.y + threadIdx.y;
    long xAdd = blockDim.x * gridDim.x;
    long yAdd = blockDim.y * gridDim.y;
    
    if(xCoord == 0 or xCoord == xSize-1)
    {
        differenceImage[xSize * yCoord + xCoord] = grayscaleImage[xSize * yCoord + xCoord];
        xCoord += xAdd;
    }
    if(yCoord == 0 or xCoord == xSize-1)
    {
        differenceImage[xSize * yCoord + xCoord] = grayscaleImage[xSize * yCoord + xCoord];
        yCoord += yAdd;
    }

    unsigned char brightestPoint;
    unsigned char darkestPoint;
    for(long x=xCoord; x < xSize; x+=xAdd)
    {   
        for(long y=yCoord; y < ySize; y+=yAdd)
        {
            if(y == 0 or x == xSize-1)
            {
                differenceImage[xSize * y + x] = grayscaleImage[xSize * y + x];
                break;
            }

            brightestPoint = 0;
            darkestPoint = 255;
            for(int i=-1; i < 2; i++)
            {
                for(int j=-1; j < 2; j++)
                {
                    if(i == 0 and j == 0) continue;
                    int currentIndex = xSize * (y + j) + (x + i);
                    brightestPoint = (brightestPoint < grayscaleImage[currentIndex])?grayscaleImage[currentIndex]:brightestPoint;
                    darkestPoint = (darkestPoint > grayscaleImage[currentIndex])?grayscaleImage[currentIndex]:darkestPoint;
                }             
            }
            differenceImage[xSize * y + x] = brightestPoint - darkestPoint;
	    }
    }
}

__device__ bool checkIfHasFloodedNeighbour(bool* isPixelFlooded, long xSize, long ySize, long xCoord, long yCoord)
{
	int xStart=-1;
	int xEnd=1;
	int yStart=-1;
	int yEnd=1;

	if(xCoord == 0) xStart = 0;
	if(xCoord == xSize-1) xEnd = 0;

	if(yCoord == 0) yStart = 0;
	if(yCoord == ySize-1) yEnd = 0;

	for(int x=xStart; x <= xEnd; x++)
	{
		for(int y=yStart; y <= yEnd; y++)
		{
			if(xStart==0 and yStart==0) continue;
			if(isPixelFlooded[xSize * (yCoord + y) + (xCoord + x)]) return true;
		}
	}
	return false;
}

__global__ void makeWatershade(unsigned char* differenceImage, bool* isPixelFlooded, long xSize, long ySize, bool* hasChangedAnyPixel, int treshold)
{
    long xCoord = blockIdx.x * blockDim.x + threadIdx.x;
    long yCoord = blockIdx.y * blockDim.y + threadIdx.y;
    long xAdd = blockDim.x * gridDim.x;
    long yAdd = blockDim.y * gridDim.y;

    for(long x=xCoord; x < xSize; x+=xAdd)
    {
        for(long y=yCoord; y < ySize; y+=yAdd)
        {
        	if(not isPixelFlooded[xSize*y+x] and differenceImage[xSize*yCoord+xCoord] <= treshold)
        	{
        		if(checkIfHasFloodedNeighbour(isPixelFlooded, xSize, ySize, x, y))
        		{
        			isPixelFlooded[xSize*y+x] = true;
        			*hasChangedAnyPixel = true;
        		}
        	}
	    }
    }

}

__global__ void makeGaussianBlur(unsigned char* imageIn, unsigned char* imageOut, long xSize, long ySize)
{
    int gaussianMask[5][5] = {{1, 1, 2, 1, 1},
                              {1, 2, 4, 2, 1},
                              {2, 4, 8, 4, 2},
                              {1, 2, 4, 2, 1},
                              {1, 1, 2, 1, 1}};
    int maskSum = 52;

    long xCoord = blockIdx.x * blockDim.x + threadIdx.x;
    long yCoord = blockIdx.y * blockDim.y + threadIdx.y;
    long xAdd = blockDim.x * gridDim.x;
    long yAdd = blockDim.y * gridDim.y;

    while(xCoord <= 1)
    {
        xCoord += xAdd;
    }
    while(yCoord <= 1)
    {
        yCoord += yAdd;
    }
    
    long inputIndex, outputIndex;
    unsigned char currentColor;
    
    //blur grayscale image
    for(long x=xCoord; x < xSize-2; x+=xAdd)
    {
        for(long y=yCoord; y < ySize-2; y+=yAdd)
        {
            currentColor = 0;
            //blur single pixel
            for (int i = 0; i<5; i++)
            {
		        for (int j = 0; j<5; j++)
                {
			        inputIndex = xSize*(y + i - 2) + (x + j - 2);
			        currentColor += imageIn[inputIndex] * gaussianMask[j][i];
		        }
	        }
            outputIndex = xSize * y + x;
            imageOut[outputIndex] = currentColor / maskSum;
        }
    }
}

__global__ void markFloodedPixels(unsigned char* imageIn, bool* isFlooded, long xSize, long ySize)
{
    long xCoord = blockIdx.x * blockDim.x + threadIdx.x;
    long yCoord = blockIdx.y * blockDim.y + threadIdx.y;
    long xAdd = blockDim.x * gridDim.x;
    long yAdd = blockDim.y * gridDim.y;

    unsigned char red = 0;
    unsigned char green = 166;
    unsigned char blue = 147;

    int pixelIndex;
    for(long x=xCoord; x < xSize; x+=xAdd)
    {
    	yCoord = blockIdx.y * blockDim.y + threadIdx.y;
        for(long y=yCoord; y < ySize; y+=yAdd)
        {
            if(isFlooded[xSize * y + x])
            {
            	pixelIndex = xSize * y * 3 + x * 3;
            	imageIn[pixelIndex + 2] = red;
            	imageIn[pixelIndex + 1] = green;
            	imageIn[pixelIndex] = blue;
            }
	    }
    }
}

__global__ void markStartPixel(bool* isFlooded, long xSize, long x, long y)
{
	isFlooded[xSize*y+x] = true;
}


Coords ImageManager::readImage(string imagePath)
{
    inputImage = imread(imagePath, CV_LOAD_IMAGE_COLOR);
    if (not inputImage.data)
    {
	    cout << "[ERROR] Could not open input image" << endl;
	    exit(-1);
    }
    Coords readImgData{inputImage.cols, inputImage.rows};
    return readImgData;
}

void ImageManager::performWatershade()
{
	outputImage = Mat(inputImage.rows, inputImage.cols, CV_8UC3);
    long cudaGrayscaleImageSize = sizeof(unsigned char) * inputImage.rows * inputImage.cols;
    long cudaInputImageSize = cudaGrayscaleImageSize * 3;
    dim3 block(blockSize, blockSize);
    dim3 grid(gridSize, gridSize);

    hipEvent_t startTime, stopTime;

    hipEventCreate(&startTime);
    hipEventCreate(&stopTime);
    hipEventRecord(startTime, 0);

    //allocate memory on GPU
    unsigned char* cudaInputImage; 
    HANDLE_ERROR(hipMalloc((void**)& cudaInputImage, cudaInputImageSize));
    HANDLE_ERROR(hipMemcpy(cudaInputImage, inputImage.data, cudaInputImageSize, hipMemcpyHostToDevice));
    
    unsigned char* cudaGrayscaleImage;
    HANDLE_ERROR(hipMalloc((void**)& cudaGrayscaleImage, cudaGrayscaleImageSize));

    //Make grayscale from image
    makeGrayscaleImage << < grid, block >> > (cudaInputImage, cudaGrayscaleImage, inputImage.cols, inputImage.rows);
    
    hipDeviceSynchronize();
    
    unsigned char* cudaBlurredImage;
    HANDLE_ERROR(hipMalloc((void**)& cudaBlurredImage, cudaGrayscaleImageSize));
    
    //Perform Gaussian blur
    makeGaussianBlur << < grid, block >> > (cudaGrayscaleImage, cudaBlurredImage, inputImage.cols, inputImage.rows);
    
    hipDeviceSynchronize();

    HANDLE_ERROR(hipFree(cudaGrayscaleImage));
    unsigned char* cudaDifferenceImage;
    HANDLE_ERROR(hipMalloc((void**)& cudaDifferenceImage, cudaGrayscaleImageSize));
    
    //Make difference image (dilatated image - eroded image)
    makeDifferenceImage<< < grid, block >> >(cudaBlurredImage, cudaDifferenceImage, inputImage.cols, inputImage.rows);
    
    hipDeviceSynchronize();

    HANDLE_ERROR(hipFree(cudaBlurredImage));

    bool* isFlooded;
    long cudaBooleanImageSize = sizeof(bool) * inputImage.rows * inputImage.cols;
    HANDLE_ERROR(hipMalloc((void**)& isFlooded, cudaBooleanImageSize));

    bool* hasAnyPixelBeenChanged;
    HANDLE_ERROR(hipMalloc((void**)& hasAnyPixelBeenChanged, sizeof(bool)));

    bool hostHasBeenChanged;
    HANDLE_ERROR(hipMemset(isFlooded, 0, cudaBooleanImageSize));
    markStartPixel<<<1,1>>>(isFlooded, inputImage.cols, startPixel.width, startPixel.heigth);
    hipDeviceSynchronize();

    int iteration = 0;
    do
    {
    	cout << "Iteracja: " << iteration++ << endl;
    	hostHasBeenChanged = false;
    	HANDLE_ERROR(hipMemcpy(hasAnyPixelBeenChanged, &hostHasBeenChanged, sizeof(bool), hipMemcpyHostToDevice));
    	makeWatershade<< < grid, block >> >(cudaDifferenceImage, isFlooded, inputImage.cols, inputImage.rows, hasAnyPixelBeenChanged, treshold);
    	hipDeviceSynchronize();
    	HANDLE_ERROR(hipMemcpy(&hostHasBeenChanged, hasAnyPixelBeenChanged, sizeof(bool), hipMemcpyDeviceToHost));
    }while(hostHasBeenChanged);

    cout << "After flooding" << endl;
	HANDLE_ERROR(hipFree(cudaDifferenceImage));

	markFloodedPixels<< < grid, block >> >(cudaInputImage, isFlooded, inputImage.cols, inputImage.rows);
    hipDeviceSynchronize();
    hipEventRecord(stopTime, 0);
    hipEventSynchronize(stopTime);
    hipEventElapsedTime(&runningTime, startTime, stopTime);


    HANDLE_ERROR(hipMemcpy(outputImage.data, cudaInputImage, cudaInputImageSize, hipMemcpyDeviceToHost));

    hipHostFree(hasAnyPixelBeenChanged);
    hipFree(cudaInputImage);
    hipFree(isFlooded);
}

void ImageManager::saveImage(string outputPath)
{
    try
    {
	    imwrite(outputPath, outputImage);
    }
    catch (Exception &e)
    {
	    cout << "[ERROR] Could not save image" << endl << e.msg;
	    exit(-1);
    } 
}

void ImageManager::setBlockSize(int blockSize)
{
    this->blockSize = blockSize;
}

int ImageManager::getBlockSize()
{
    return this->blockSize;
}

void ImageManager::setGridSize(int gridSize)
{
    this->gridSize = gridSize;
}

int ImageManager::getGridSize()
{
    return this->gridSize;
}

void ImageManager::setStartPixel(Coords startCoords)
{
    startPixel = startCoords;
}

void ImageManager::setTreshold(int treshold)
{
    this->treshold = treshold;
}

void ImageManager::clearRunningTime()
{
    runningTime = 0.0;
}
double ImageManager::getRunningTime()
{
    return runningTime;
}

void validateArguments(int argc, char** argv, string& inputPath, string& outputPath)
{
	if (argc < 3)
	{
		cout << "[ERR] Wrong count of parameters!" << endl;
		exit(-1);
	}
	inputPath = argv[1];
	outputPath = argv[2];
}

int main(int argc, char** argv) {
	Mat inputImage;
	string inputPath, outputPath;
	validateArguments(argc, argv, inputPath, outputPath);

    unique_ptr<ImageManager> imageManager;
    imageManager = make_unique<ImageManager>();
    

    Coords imgSize = imageManager->readImage(inputPath);

 /*
    Coords startCoords{0, 0};
    cout << "Rozmiar obrazka (szer, wys): " << imgSize.width << ", " << imgSize.heigth << ". Podaj piksel startowy (x, y): ";
    cin >> startCoords.width;
    cin >> startCoords.heigth;
    imageManager->setStartPixel(startCoords);

    int treshold;
    cout << "Podaj treshold zalania (0-255): ";
    cin >> treshold;
    imageManager->setTreshold(treshold);

	int size;
    cout << "Podaj rozmiar bloku (dlugosc jednego boku, rzeczywisty rozmiar bedzie wynosil kwadrat z podanej liczby): ";
    cin >> size;
    imageManager->setBlockSize(size);
    
    cout << "Podaj rozmiar gridu (dlugosc jednego boku, rzeczywisty rozmiar bedzie wynosil kwadrat z podanej liczby): ";
    cin >> size;
    imageManager->setGridSize(size);
    
 */
    Coords startCoords{100, 100};
    imageManager->setStartPixel(startCoords);
    imageManager->setTreshold(0);
    imageManager->setBlockSize(10);
    imageManager->setGridSize(10);

    cout << "grid size, block size, time" << endl;


    imageManager->clearRunningTime();
    imageManager->performWatershade();
    double time = imageManager->getRunningTime();
    cout << imageManager->getGridSize() * imageManager->getGridSize() << ", " << imageManager->getBlockSize() * imageManager->getBlockSize() << ", " << imageManager->getRunningTime();
    imageManager->saveImage(outputPath);


	return 0;
}
